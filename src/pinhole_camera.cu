#include "hip/hip_runtime.h"

/*
 * Copyright (c) 2008 - 2009 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and proprietary
 * rights in and to this software, related documentation and any modifications thereto.
 * Any use, reproduction, disclosure or distribution of this software and related
 * documentation without an express license agreement from NVIDIA Corporation is strictly
 * prohibited.
 *
 * TO THE MAXIMUM EXTENT PERMITTED BY APPLICABLE LAW, THIS SOFTWARE IS PROVIDED *AS IS*
 * AND NVIDIA AND ITS SUPPLIERS DISCLAIM ALL WARRANTIES, EITHER EXPRESS OR IMPLIED,
 * INCLUDING, BUT NOT LIMITED TO, IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A
 * PARTICULAR PURPOSE.  IN NO EVENT SHALL NVIDIA OR ITS SUPPLIERS BE LIABLE FOR ANY
 * SPECIAL, INCIDENTAL, INDIRECT, OR CONSEQUENTIAL DAMAGES WHATSOEVER (INCLUDING, WITHOUT
 * LIMITATION, DAMAGES FOR LOSS OF BUSINESS PROFITS, BUSINESS INTERRUPTION, LOSS OF
 * BUSINESS INFORMATION, OR ANY OTHER PECUNIARY LOSS) ARISING OUT OF THE USE OF OR
 * INABILITY TO USE THIS SOFTWARE, EVEN IF NVIDIA HAS BEEN ADVISED OF THE POSSIBILITY OF
 * SUCH DAMAGES
 */

#include <optix_world.h>
#include "helpers.h"
#include "paint_camera.h"
#include <vector>
//#include <optixpp_namespace.h>

using namespace optix;
using namespace std;

struct PerRayData_radiance
{
  float3 result;
  float  importance;
  int    depth;
  void*   shaderIdx;
};

rtBuffer<void*, 2>        shader_buffer;
rtBuffer<uint>          printEnabled;

rtDeclareVariable(float3,        eye, , );
rtDeclareVariable(float3,        U, , );
rtDeclareVariable(float3,        V, , );
rtDeclareVariable(float3,        W, , );
rtDeclareVariable(float3,        bad_color, , );
rtDeclareVariable(float,         scene_epsilon, , );
rtBuffer<uchar4, 2>              output_buffer;
rtDeclareVariable(rtObject,      top_object, , );
rtDeclareVariable(unsigned int,  radiance_ray_type, , );

rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtDeclareVariable(uint2, launch_dim,   rtLaunchDim, );
rtDeclareVariable(float, time_view_scale, , ) = 1e-6f;

rtDeclareVariable(unsigned int,  paint_camera_type, , );

rtDeclareVariable(uint2, pick_index, , );
//rtDeclareVariable(int, printEnabled, , ) = 0;
//#define TIME_VIEW

RT_PROGRAM void pinhole_camera()
{
#ifdef TIME_VIEW
  clock_t t0 = clock(); 
#endif
  float2 d = make_float2(launch_index) / make_float2(launch_dim) * 2.f - 1.f;
  float3 ray_origin = eye;
  float3 ray_direction = normalize(d.x*U + d.y*V + W);

  // Posing camera
  if ( paint_camera_type == 1 || paint_camera_type == 2 )
    ray_direction = normalize( ray_direction + cameraTexture(d, camera_pose_map) );

  optix::Ray ray = optix::make_Ray(ray_origin, ray_direction, radiance_ray_type, scene_epsilon, RT_DEFAULT_MAX);

  // Painting camera
  if ( paint_camera_type == 0 || paint_camera_type == 2)
    paint_camera( &ray );
//  else if ( paint_camera_type == 1 )
//    pose_camera( &ray );
//  else{
//      paint_camera( &ray );
//      pose_camera( &ray );
//  }

  PerRayData_radiance prd;
  prd.importance = 1.f;
  prd.depth = 0;
  prd.shaderIdx = NULL;

  rtTrace(top_object, ray, prd);
  //rtPrintf( "d: (%f, %f)\n", d.x, d.y );

#ifdef TIME_VIEW
  clock_t t1 = clock(); 
 
  float expected_fps   = 1.0f;
  float pixel_time     = ( t1 - t0 ) * time_view_scale * expected_fps;
  output_buffer[launch_index] = make_color( make_float3(  pixel_time ) ); 
#else
  output_buffer[launch_index] = make_color( prd.result );
  shader_buffer[launch_index] = prd.shaderIdx;


  // debug rtPrintf
  if (launch_index.x >= pick_index.x && launch_index.y >= pick_index.y && printEnabled[0]){
      //rtPrintf( "pick_index: (%u, %u)\n", pick_index.x, pick_index.y );
      //rtPrintf( "Color: (%u, %u, %u)\n", output_buffer[launch_index].x,
      //          output_buffer[launch_index].y,
      //          output_buffer[launch_index].z );
      //rtPrintf( "shader Index: %u\n", shader_buffer[launch_index] );
      printEnabled[0] = false;
  }

#endif
}



RT_PROGRAM void exception()
{
  const unsigned int code = rtGetExceptionCode();
  rtPrintf( "Caught exception 0x%X at launch index (%d,%d)\n", code, launch_index.x, launch_index.y );
  output_buffer[launch_index] = make_color( bad_color );
}
