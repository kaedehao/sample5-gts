#include "hip/hip_runtime.h"
// Environment map background

#include <optix.h>
#include <optixu/optixu_math_namespace.h>

using namespace optix;

struct PerRayData_radiance
{
  float3 result;
  float importance;
  int depth;
};

rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );

rtTextureSampler<float4, 2> envmap;

RT_PROGRAM void envmap_miss()
{
  float theta = atan2f( ray.direction.x, ray.direction.z );
  float phi   = M_PIf * 0.5f -  acosf( ray.direction.y );
  float u     = (theta + M_PIf) * (0.5f * M_1_PIf);
  float v     = 0.5f * ( 1.0f + sin(phi) );
  prd_radiance.result = make_float3( tex2D(envmap, u, v) );
  //rtPrintf( "Environment texture color: %d, %d, %d!\n", prd_radiance.result.x, prd_radiance.result.y, prd_radiance.result.z );
}
